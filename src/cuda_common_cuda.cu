#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.hpp"

//hipError_t err;

//extern "c"
void _cuda_safe_mem (hipError_t err, const char *file, unsigned int line){

  if(hipSuccess != err) {
    fprintf(stderr, "Cuda Memory error at %s:%u.\n", file, line);
    printf("CUDA error: %s\n", hipGetErrorString(err));
    if (err == hipErrorInvalidValue)
      fprintf(stderr, "You may have tried to allocate zero memory at %s:%u.\n", file, line);
    //errexit();
    exit(1);
  } else {
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error found during memory operation. Possibly however from an failed operation before. %s:%u.\n", file, line);
      printf("CUDA error: %s\n", hipGetErrorString(err));
      if(err == hipErrorInvalidValue)
        fprintf(stderr, "You may have tried to allocate zero memory before %s:%u.\n", file, line);
      //errexit();
      exit(1);
    }
  }
}

//extern "c"
void _cuda_check_errors (const dim3 &block, const dim3 &grid, const char *function, const char *file, unsigned int line) {

  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf (stderr, "error \"%s\" calling %s with dim %d %d %d, grid %d %d %d in %s:%u\n", 
             hipGetErrorString(err), function, block.x, block.y, block.z, grid.x, grid.y, grid.z, file, line);
    //errexit();
    exit(1);
  }
}

